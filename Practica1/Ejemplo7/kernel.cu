#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>

#include "kernel.h"



void cannyGPU(float *im, float *image_out,
	float level,
	int height, int width)
{
	// GPU Memory
	float *imageBWGPU;	hipMalloc((void**)&imageBWGPU, sizeof(float)*width*height);
	hipMemcpy(imageBWGPU, im, sizeof(float)*width*height, hipMemcpyHostToDevice);
	float *NRGPU;		hipMalloc((void**)&NRGPU, sizeof(float)*width*height);
	float *GGPU;		hipMalloc((void**)&GGPU, sizeof(float)*width*height);
	float *phiGPU;		hipMalloc((void**)&phiGPU, sizeof(float)*width*height);
	float *GxGPU;		hipMalloc((void**)&GxGPU, sizeof(float)*width*height);
	float *GyGPU;		hipMalloc((void**)&GyGPU, sizeof(float)*width*height);
	int *pedgeGPU;		hipMalloc((void**)&pedgeGPU, sizeof(int)*width*height);
	float *imageOUTGPU;	hipMalloc((void**)&imageOUTGPU, sizeof(float)*width*height);
	
	dim3 dimBlock(4, 4);
	dim3 dimGrid(height-4/16, width-4/16);

	NRGPU<<<dimGrid,dimBlock>>>(height, width, NRGPU, imageBWGPU);
	

	// Copy results to output
	hipMemcpy(image_out, imageOUTGPU, sizeof(float)*width*height, hipMemcpyDeviceToHost);
}

__global__ void NRGPU(int height, int width, float *NR, float *im)
{
	int i, j;
	i = blockIdx.x + ((threadIdx.x/(width-4)) + 2);
	for(i=2; i<height-2; i++)
		for(j=2; j<width-2; j++)
		{
			// Noise reduction
			NR[i*width+j] =
				 (2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
				+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
				+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
				+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
				+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
				/159.0;
		}

}
